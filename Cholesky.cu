
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <ctime>
#include <iostream>
#include <cstdlib> 
using namespace std; 
  
const int MAX = 100; 
 

__global__ void Cholesky_GPU(double *matrix, int n){
	
	//n threads running in parallel

	//int x = blockIdx.x;
	int y = threadIdx.x;
	//int i = x;
	int j = y;
	
	extern __device__ __shared__ double localMatrix[];
//	extern __device__ __shared__ double sum[];
	//matrix2d[x][y] = matrix1d[x*n+y]	
	
	//Copy to shared mem

	for(int i=0; i<n; i++)
		localMatrix[i*n+j] = matrix[i*n+j];

	localMatrix[n*n+j] = 0; // sum column
	__syncthreads();

	//Do the calc;
	#pragma unroll
	for(int i=0; i<n; i++){
		if(j<i){
			localMatrix[i*n+j] = 0;
		}
		if(j>=i) {
			localMatrix[n*n+j]=0;//initialize sum to 0
			for(int k=0; k<i; k++)
				localMatrix[n*n+j] +=localMatrix[k*n+i]*localMatrix[k*n+j]; // sums
			//if(j<i){
			//	localMatrix[i*n+j]=0;
			//}
			if(i == j){
				localMatrix[i*n+j] = sqrt(localMatrix[i*n+j] - localMatrix[n*n+j]);
			}if(j > i){
				localMatrix[i*n+j] = (localMatrix[i*n+j] - localMatrix[n*n+j])/localMatrix[i*n+i];
			}
		}
	}


	__syncthreads();

	
	for(int i=0; i<n; i++)
		matrix[i*n+j] = localMatrix[i*n+j];
	//Copy back



}

int main(){
	srand(time(0));
    float totalTime = 0;
    int n = 100;  //Matrix size
    //cout << n + "\n";
    double matrix1[MAX][MAX];
    double matrix2[MAX][MAX];
    for(int a = 0; a<5000; a++){
    	for(int j = 0; j<n; j++){
    		for(int k = 0; k<j+1; k++){
    			matrix1[k][j] = (rand()%100) + 1;
                matrix2[j][k] = matrix1[k][j];
    			//cout << matrix[j][k] + "\n";
    		}
            for(int k = 0; k<j; k++){
                matrix1[j][k] = 0;
                matrix2[k][j] = 0;
            }
    	}
		/*
        for (int i = 0; i < n; i++) {
            for (int j = 0; j < n; j++) 
                cout << setw(6) << matrix1[i][j] << " ";  
            cout << endl;
            }
            cout << "__________________________________________________\n";
        for (int i = 0; i < n; i++) { 
            for (int j = 0; j < n; j++) 
                cout << setw(6) << matrix2[i][j] << " ";  
            cout << endl;
            }
            cout << "__________________________________________________\n";
        */
		double matrix[MAX][MAX];
        for(int i = 0; i < n; ++i){
            for(int j = 0; j < n; ++j){
                for(int k = 0; k < n; ++k){
                    //cout << matrix1[i][k]<<"*"<<matrix2[k][j]<<"+";
					matrix[i][j] += matrix2[i][k] * matrix1[k][j];
                }
				//cout<<endl;
			}
		}
		//matrix={{2,-1,0},{-1,2,0},{0,-1,2}};
        /*
        for (int i = 0; i < n; i++) { 
          for (int j = 0; j < n; j++) 
            cout << setw(6) << matrix[i][j] << " ";  
        cout << endl;
        }
        cout << "__________________________________________________\n";
        */
		double *matrix1d = (double*)malloc(n*n*sizeof(double));
		
        for (int i = 0; i < n; i++) { 
        	for (int j = 0; j < n; j++){
        		matrix1d[i*n+j] = matrix[i][j];  
		}
        //cout << endl;
        }
		/*
		for(int i=0; i<n; i++){
			for(int j=0; j<n; j++)
				cout<<matrix1d[i*n+j] << "  ";
		}
		cout<<endl;
		*/
		//Cuda 
		float time;
		hipEvent_t start_kernel, stop_kernel;
		hipEventCreate(&start_kernel);
		hipEventCreate(&stop_kernel);
		//cudaEventRecord(start_kernel, 0);

		double *dev_matrix ;
		hipMalloc(&dev_matrix,n*n*sizeof(double));
		hipMemcpy(dev_matrix, matrix1d, n*n*sizeof(double), hipMemcpyHostToDevice);

		dim3 dimGrid(1,1,1);
		dim3 dimBlock(n,1,1);
		
		hipEventRecord(start_kernel,0);
		Cholesky_GPU<<<dimGrid, dimBlock, n*(n+1)>>>(dev_matrix, n);
		hipEventRecord(stop_kernel,0);
		hipEventSynchronize(stop_kernel);
		hipEventElapsedTime(&time, start_kernel, stop_kernel);	
		hipDeviceSynchronize();
		totalTime+=time;
		hipMemcpy(matrix1d, dev_matrix, n*n*sizeof(double), hipMemcpyDeviceToHost);
		/*
		for(int i=0; i<n; i++){
			for(int j=0; j<n; j++){
				cout<<matrix1d[i*n+j]<<" ";
			}
			cout<<endl;
		}
		*/

    	//Cholesky_CPU(matrix,n);
    	//Cholesky_CPU_optimized(matrix,n);
    }
	totalTime=totalTime/1000.0;
    printf("%f\n", totalTime);
	//cout << totalTime;
    //cout << endl;
    return 0; 
}
